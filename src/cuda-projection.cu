#include "hip/hip_runtime.h"
/**
 * @file cuda-projection.cu
 * @author Enrico Marchionni (enrico.marchionni@studio.unibo.it)
 * @brief A CUDA implementation of the Siddon's projection algorithm.
 * @date 2025-02
 * @details
 * This file contains an implementation of the projection algorithm
 * for generating 2D projections of a 3D object.
 * The algorithm is based on Siddon's algorithm and is parallelized
 * using CUDA.
 * The algorithm reads the 3D object from a file and writes the
 * reconstructed projection images to another file.
 *
 * The algorithm is divided into several functions, each of which
 * implements some steps from Siddon's algorithm.
 * The main function reads the 3D image from the file and computes
 * the 2D projections.
 * @copyright
 * ```text
 * This file is part of 3D-CT-projection-cuda
 * (https://github.com/EnryMarch10/3D-CT-projection-cuda).
 * Copyright (C) 2025 Enrico Marchionni
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 * ```
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include "hpc.h"
#include "common.h"

/************************************************* DEVICE *************************************************/

// Inputs configuration limits
// 1024^3 (voxels cube size) * 8 (double size in Bytes) = 8 GiB
// max file size if the voxels are the same for each axis, and if the doubles are 8 Bytes
#define MAX_PLANES 1300u
#define MAX_PLANES_x2 (MAX_PLANES * 2)
#define MAX_PLANES_x3 (MAX_PLANES * 3)
// CUDA devices typically have 64 KB of constant memory
// This solution allocates 16KB of memory considering the 2 tables + other variables of small size
#define MAX_CONSTANT_MEMORY 8192u // 8 KB in Bytes
#define MAX_TABLES_SIZE (MAX_CONSTANT_MEMORY / sizeof(double)) // 8 KB in doubles
// Cuda limits
#define BLKDIM_STEP 16u // Max is usually 32, but for some GPUs the amount of registers is not enough in that case
#define BLKDIM (BLKDIM_STEP * BLKDIM_STEP)

__constant__ unsigned short d_pixelDim;
__constant__ unsigned short d_angularTrajectory;
__constant__ unsigned short d_positionsAngularDistance;
__constant__ unsigned short d_voxelXDim;
__constant__ unsigned short d_voxelYDim;
__constant__ unsigned short d_voxelZDim;

__constant__ unsigned d_objectSideLength;
__constant__ unsigned d_detectorSideLength;
__constant__ unsigned d_distanceObjectDetector;
__constant__ unsigned d_distanceObjectSource;

__constant__ unsigned short d_nVoxel[3];
__constant__ unsigned short d_nPlanes[3];

__constant__ double d_gl_sinTable[MAX_TABLES_SIZE];
__constant__ double d_gl_cosTable[MAX_TABLES_SIZE];

__device__ unsigned short d_yVoxels;

__device__ double d_gMin;
__device__ double d_gMax;

/************************************************* HOST *************************************************/

unsigned short yVoxels = 0;

unsigned short gl_pixelDim;
unsigned short gl_angularTrajectory;
unsigned short gl_positionsAngularDistance;
unsigned short gl_voxelXDim;
unsigned short gl_voxelYDim;
unsigned short gl_voxelZDim;
unsigned short gl_nVoxel[3];
unsigned short gl_nPlanes[3];

unsigned gl_objectSideLength;
unsigned gl_detectorSideLength;
unsigned gl_distanceObjectDetector;
unsigned gl_distanceObjectSource;

double *gl_sinTable, *gl_cosTable;

double *d_f, *d_g;

/**
 * @brief Initializes `sin` and `cos` tables, with default values for a certain length.
 *
 * @param sinTable An array containing a certain number of sin values to precalculate.
 * @param cosTable An array containing a certain number of cos values to precalculate.
 * @param length The length of the arrays.
 */
void initTables(double *const sinTable, double *const cosTable, const unsigned short length)
{
    for (unsigned short positionIndex = 0; positionIndex < length; positionIndex++) {
        sinTable[positionIndex] = sin((-(double) gl_angularTrajectory / 2 + (double) positionIndex * gl_positionsAngularDistance) * M_PI / 180);
        cosTable[positionIndex] = cos((-(double) gl_angularTrajectory / 2 + (double) positionIndex * gl_positionsAngularDistance) * M_PI / 180);
    }
}

/**
 * @brief Computes the coordinate of a plane parallel relative to the YZ plane.
 *
 * @param index It is the index of the plane to be returned where 0 is the index of the smallest-valued coordinate plane.
 * @return The coordinate of a plane parallel relative to the YZ plane.
 * @return __device__ Indicates that this is a CUDA function that can be called from a kernel.
 */
__device__ double getXPlane(const unsigned short index)
{
    return -(double) d_objectSideLength / 2 + (double) index * d_voxelXDim;
}

/**
 * @brief Computes the coordinate of a plane parallel relative to the XZ plane.
 *
 * @param index It is the index of the plane to be returned where 0 is the index of the smallest-valued coordinate plane.
 * @return The coordinate of a plane parallel relative to the XZ plane.
 * @return __device__ Indicates that this is a CUDA function that can be called from a kernel.
 */
__device__ double getYPlane(const unsigned short index)
{
    return -(double) d_objectSideLength / 2 + (double) index * d_voxelYDim;
}

/**
 * @brief Computes the coordinate of a plane parallel relative to the XY plane.
 *
 * @param index It is the index of the plane to be returned where 0 is the index of the smallest-valued coordinate plane.
 * @return The coordinate of a plane parallel relative to the XY plane.
 * @return __device__ Indicates that this is a CUDA function that can be called from a kernel.
 */
__device__ double getZPlane(const unsigned short index)
{
    return -(double) d_objectSideLength / 2 + (double) index * d_voxelZDim;
}

/**
 * @brief Computes the maximum parametric value a, representing the last intersection between ray and object.
 *
 * @param a It is the array containing the parametric value of the intersection between the ray and the object's side along each axis.
 * @param isParallel It is a value corresponding to the axis to which the array is orthogonal, -1 otherwise.
 * @return The maximum parametric value a, representing the last intersection between ray and object.
 * @return __device__ Indicates that this is a CUDA function that can be called from a kernel.
 */
__device__ double getAMax(double a[3][2], const char isParallel)
{
    double tempMax[3];
    double aMax = 1;
    for (char i = 0; i < 3; i++) {
        if (i != isParallel) {
            tempMax[i] = a[i][0] > a[i][1] ? a[i][0] : a[i][1];
        }
    }
    for (char i = 0; i < 3; i++) {
        if (i != isParallel) {
            aMax = aMax < tempMax[i] ? aMax : tempMax[i];
        }
    }
    return aMax;
}

/**
 * @brief Computes the minimum parametric value a, representing the fist intersection between ray and object.
 *
 * @param a It is the array containing the parametric value of the intersection between the ray and the object's side along each axis.
 * @param isParallel It is a value corresponding to the axis to which the array is orthogonal, -1 otherwise.
 * @return The minimum parametric value a, representing the first intersection between ray and object.
 * @return __device__ Indicates that this is a CUDA function that can be called from a kernel.
 */
__device__ double getAMin(double a[3][2], const char isParallel)
{
    double tempMin[3];
    double aMin = 0;
    for (char i = 0; i < 3; i++) {
        if (i != isParallel) {
            tempMin[i] = a[i][0] < a[i][1] ? a[i][0] : a[i][1];
        }
    }
    for (char i = 0; i < 3; i++) {
        if (i != isParallel) {
            aMin = aMin > tempMin[i] ? aMin : tempMin[i];
        }
    }
    return aMin;
}

/**
 * @brief Computes the the intersections between a ray and a set of planes.
 *
 * @param source Represents the coordinate of the source.
 * @param pixel Represents the coordinate of a unit of the detector, relative to the specified source.
 * @param planes It is an array that contains the coordinates of each plane.
 * @param nPlanes Specifies the number of planes.
 * @param a It is an array that will be filled with the parametric values that identify the intersection points between the
 * ray and each plane.
 * @return 0 if ray is parallel to the planes, 1 otherwise.
 * @return __device__ Indicates that this is a CUDA function that can be called from a kernel.
 */
__device__ char getIntersection(const double source, const double pixel, const double *const planes, const unsigned short nPlanes, double *const a)
{
    if (source - pixel != 0) {
        for (unsigned short i = 0; i < nPlanes; i++) {
            a[i] = (planes[i] - source) / (pixel - source);
        }
        return 1;
    }
    return 0;
}

/**
 * @brief Computes the coordinates of the planes necessary to compute the intersections with the ray.
 * Then it calls \ref getIntersection().
 *
 * @param source Represents the coordinate of the source.
 * @param pixel Represents the coordinate of a unit of the detector, relative to the specified source.
 * @param planeIndexesRanges It is a structure containing the index ranges for planes.
 * @param a It is an array that will be filled with the parametric values that identify the intersection points between the
 * ray and each plane.
 * @param axis It is the axis orthogonal to the set of planes to which compute the intersection.
 * @return __device__ Indicates that this is a CUDA function that can be called from a kernel.
 */
__device__ void getAllIntersections(const double source, const double pixel, const Ranges planeIndexesRanges, double *const a, const Axis axis)
{
    int start = 0, end = 0;
    double d;

    start = planeIndexesRanges.minIndx;
    end = planeIndexesRanges.maxIndx;
    if (end > start) { // Avoids management of invalid array
        assert(end - start <= MAX_PLANES);
        double plane[MAX_PLANES];
        if (axis == X) {
            plane[0] = getXPlane(start);
            d = d_voxelXDim;
            if (pixel - source < 0) {
                plane[0] = getXPlane(end);
                d = -(double) d_voxelXDim;
            }
        } else if (axis == Y) {
            plane[0] = getYPlane(start);
            d = d_voxelYDim;
            if (pixel - source < 0) {
                plane[0] = getYPlane(end);
                d = -(double) d_voxelYDim;
            }
        } else /* if (axis == Z) */ {
            plane[0] = getZPlane(start);
            d = d_voxelZDim;
            if (pixel - source < 0) {
                plane[0] = getZPlane(end);
                d = -(double) d_voxelZDim;
            }
        }

        for (unsigned short i = 1; i < end - start; i++) {
            plane[i] = plane[i - 1] + d;
        }
        getIntersection(source, pixel, plane, end - start, a);
    }
}

/**
 * @brief Retrieves the range of parametric values of the planes.
 *
 * @param source Represents the coordinate of the source.
 * @param pixel Represents the coordinate of a unit of the detector, relative to the specified source.
 * @param isParallel It has a value corresponding to the axis to which the array is orthogonal, -1 otherwise.
 * @param aMin It is the minimum parametric value of the intersection between the ray and the object.
 * @param aMax It is the maximum parametric value of the intersection between the ray and the object.
 * @param axis It is the axis orthogonal to the plane.
 * @return The range of parametric values of the planes.
 * @return __device__ Indicates that this is a CUDA function that can be called from a kernel.
 */
__device__ Ranges getRangeOfIndex(const double source, const double pixel, const char isParallel, const double aMin, const double aMax, const Axis axis)
{
    Ranges idxs;
    double firstPlane, lastPlane;
    unsigned short voxelDim;

    if (axis == X) {
        voxelDim = d_voxelXDim;
        firstPlane = getXPlane(0);
        lastPlane = getXPlane(d_nPlanes[X] - 1);
    } else if (axis == Y) {
        voxelDim = d_voxelYDim;
        firstPlane = getYPlane(0);
        lastPlane = getYPlane(d_nPlanes[Y] - 1);
    } else /* if (axis == Z) */ {
        voxelDim = d_voxelZDim;
        firstPlane = getZPlane(0);
        lastPlane = getZPlane(d_nPlanes[Z] - 1);
    }

    // Gets range of indexes of XZ parallel planes
    if (isParallel != Y) {
        if (pixel - source >= 0) {
            idxs.minIndx = d_nPlanes[axis] - ceil((lastPlane - aMin * (pixel - source) - source) / voxelDim);
            idxs.maxIndx = 1 + floor((aMax * (pixel - source) + source - firstPlane) / voxelDim);
        } else {
            idxs.minIndx = d_nPlanes[axis] - ceil((lastPlane - aMax * (pixel - source) - source) / voxelDim);
            idxs.maxIndx = floor((aMin * (pixel - source) + source - firstPlane) / voxelDim);
        }
    } else {
        idxs.minIndx = 0;
        idxs.maxIndx = 0;
    }
    return idxs;
}

/**
 * @brief Merges two sorted arrays into one single sorted array.
 *
 * @param a It is a pointer to a sorted array.
 * @param b It is a pointer to a sorted array.
 * @param lenA It is the length of the array `a`.
 * @param lenB It is the length of the array `b`.
 * @param c It is the computed merged array.
 * @return The length of the merged array.
 * @return __device__ Indicates that this is a CUDA function that can be called from a kernel.
 */
__device__ unsigned short merge(const double *const a, const double *const b, const unsigned short lenA, const unsigned short lenB, double *const c)
{
    unsigned short i = 0, j = 0, k = 0;
    while (j < lenA && k < lenB) {
        if (a[j] < b[k]) {
            c[i] = a[j];
            j++;
        } else {
            c[i] = b[k];
            k++;
        }
        i++;
    }
    while (j < lenA) {
        c[i] = a[j];
        i++;
        j++;
    }
    while (k < lenB) {
        c[i] = b[k];
        i++;
        k++;
    }
    return lenA + lenB;
}

/**
 * @brief Merges three sorted arrays into one single sorted array.
 *
 * @param a It is a pointer to a sorted array.
 * @param b It is a pointer to a sorted array.
 * @param c It is a pointer to a sorted array.
 * @param lenA It is the length of the array `a`.
 * @param lenB It is the length of the array `b`.
 * @param lenC It is the length of the array `c`.
 * @param merged It is the computed merged array.
 * @return The length of the merged array.
 * @return __device__ Indicates that this is a CUDA function that can be called from a kernel.
 */
__device__ unsigned short merge3(const double *const a, const double *const b, const double *const c, const unsigned short lenA, const unsigned short lenB, const unsigned short lenC, double *const merged)
{
    assert(lenA + lenB + lenC <= MAX_PLANES_x3);
    double ab[MAX_PLANES_x3];
    const unsigned short length = merge(a, b, lenA, lenB, ab);
    return merge(ab, c, length, lenC, merged);
}

/**
 * @brief Retrieves the cartesian coordinates of the source.
 *
 * @param sinTable An array containing a certain number of precalculated sin values.
 * @param cosTable An array containing a certain number of precalculated cos values.
 * @param index A value that defines the angle being considered.
 * @return The coordinates of the source.
 * @return __device__ Indicates that this is a CUDA function that can be called from a kernel.
 */
__device__ Point getSource(const double *const sinTable, const double *const cosTable, const unsigned short index)
{
    Point source;

    source.z = 0.0;
    source.x = sinTable[index] * d_distanceObjectSource;
    source.y = cosTable[index] * d_distanceObjectSource;

    return source;
}

/**
 * @brief Retrieves the cartesian coordinates of a unit of the detector.
 *
 * @param sinTable An array containing a certain number of precalculated sin values.
 * @param cosTable An array containing a certain number of precalculated cos values.
 * @param r The row of the detector matrix.
 * @param c The column of the detector matrix.
 * @param index A value that defines the angle being considered, and consequently, the source.
 * @return The coordinates of a unit of the detector, relative to the specified source.
 * @return __device__ Indicates that this is a CUDA function that can be called from a kernel.
 */
__device__ Point getPixel(const double *const sinTable, const double *const cosTable, const unsigned r, const unsigned c, const unsigned short index)
{
    Point pixel;
    const double sinAngle = sinTable[index];
    const double cosAngle = cosTable[index];
    const double elementOffset = d_detectorSideLength / 2 - d_pixelDim / 2;

    pixel.x = -(double) d_distanceObjectDetector * sinAngle + cosAngle * (-elementOffset + (double) d_pixelDim * c);
    pixel.y = -(double) d_distanceObjectDetector * cosAngle - sinAngle * (-elementOffset + (double) d_pixelDim * c);
    pixel.z = -elementOffset + (double) d_pixelDim * r;

    return pixel;
}

/**
 * @brief Computes a coordinate of the two planes of the object's sides orthogonal to the x axis.
 *
 * @param planes It is a pointer to an array of two elements,
 * each one of them is the coordinate of a plane parallel relative to the YZ plane.
 */
__device__ void getSidesXPlanes(double *const planes)
{
    planes[0] = getXPlane(0);
    planes[1] = getXPlane(d_nPlanes[X] - 1);
}

/**
 * @brief Computes a coordinate of the two planes of the object's sides orthogonal to the y axis.
 *
 * @param planes It is a pointer to an array of two elements,
 * each one of them is the coordinate of a plane parallel relative to the XZ plane.
 * @param slice It is a number that indicates the first voxel in the y axis from which the projection is being computed.
 * In this case this limits the planes considered.
 * @return __device__ Indicates that this is a CUDA function that can be called from a kernel.
 */
__device__ void getSidesYPlanes(double *const planes, const unsigned short slice)
{
    planes[0] = getYPlane(slice);
    planes[1] = getYPlane(min(d_nPlanes[Y] - 1, d_yVoxels + slice));
}

/**
 * @brief Computes a coordinate of the two planes of the object's sides orthogonal to the z axis.
 *
 * @param planes It is a pointer to an array of two elements,
 * each one of them is the coordinate of a plane parallel relative to the XY plane.
 * @return __device__ Indicates that this is a CUDA function that can be called from a kernel.
 */
__device__ void getSidesZPlanes(double *const planes)
{
    planes[0] = getZPlane(0);
    planes[1] = getZPlane(d_nPlanes[Z] - 1);
}

/**
 * @brief Computes the projection attenuation of the radiological path of a ray.
 *
 * @param slice It is a number that indicates the first voxel in the y axis from which the projection is being computed.
 * @param source Represents the coordinate of the source.
 * @param pixel Represents the coordinate of the unit of the detector.
 * @param a It is an array that contains all intersection points merged, expressed parametrically.
 * @param lenA It is the length of the corresponding array.
 * @param f It is an array of the coefficients of attenuation for each voxel.
 * @return The computed projection attenuation of the radiological path of a ray.
 * @return __device__ Indicates that this is a CUDA function that can be called from a kernel.
 */
__device__ double computeAbsorption(const unsigned short slice, const Point source, const Point pixel, const double *const a, const unsigned short lenA, const double *const f)
{
    double g = 0.0;

    if (lenA > 0) { // Avoids overflow on unsigned value
        const double deltaX = pixel.x - source.x;
        const double deltaY = pixel.y - source.y;
        const double deltaZ = pixel.z - source.z;
        const double d12 = sqrt(pow(deltaX, 2) + pow(deltaY, 2) + pow(deltaZ, 2));
        for (unsigned short i = 0; i < lenA - 1; i++) {
            const double aMid = (a[i + 1] + a[i]) / 2;
            const unsigned short x = min((int) ((source.x + aMid * deltaX - getXPlane(0)) / d_voxelXDim), d_nVoxel[X] - 1);
            const unsigned short y = min((int) ((source.y + aMid * deltaY - getYPlane(slice)) / d_voxelYDim), min(d_nVoxel[Y] - 1, d_yVoxels - 1));
            const unsigned short z = min((int) ((source.z + aMid * deltaZ - getZPlane(0)) / d_voxelZDim), d_nVoxel[Z] - 1);

            // In a 3D matrix it would be: f[x][z][y]
            // d12 * (a[i + 1] - a[i] = segment length
            g += f[x + (unsigned) z*d_nVoxel[Z] + (unsigned) y*d_nVoxel[X]*d_nVoxel[Z]] * d12 * (a[i + 1] - a[i]);
        }
    }
    return g;
}

/**
 * @brief Atomically sets `addr` value to `value` if it is lower.
 *
 * @param addr The address that contains the value that could be set atomically.
 * @param value It is the value that could be exchanged atomically with `addr` value.
 * @return __device__ Indicates that this is a CUDA function that can be called from a kernel.
 */
__device__ __forceinline__ double atomicMinDouble(double *const addr, const double value) {
    unsigned long long *addr_as_ull = (unsigned long long *) addr;
    unsigned long long old = *addr_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(addr_as_ull, assumed, __double_as_longlong(fmin(__longlong_as_double(assumed), value)));
    } while (assumed != old);

    return __longlong_as_double(old);
}

/**
 * @brief Atomically sets `addr` value to `value` if it is greater.
 *
 * @param addr The address that contains the value that could be set atomically.
 * @param value It is the value that could be exchanged atomically with `addr` value.
 * @return __device__ Indicates that this is a CUDA function that can be called from a kernel.
 */
__device__ __forceinline__ double atomicMaxDouble(double *const addr, const double value) {
    unsigned long long *addr_as_ull = (unsigned long long *) addr;
    unsigned long long old = *addr_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(addr_as_ull, assumed, __double_as_longlong(fmax(__longlong_as_double(assumed), value)));
    } while (assumed != old);

    return __longlong_as_double(old);
}

/**
 * @brief Computes the projection of a sub-section of the object into the detector for each source position on an NVIDIA GPU.
 *
 * @param slice It is a number that indicates the first voxel in the y axis from which the projection is being computed.
 * @param nTheta It is the number of angular positions.
 * @param nSidePixels It is the number of pixels per size of the detector.
 * @param f It is an array that contains the coefficients of attenuation of the voxels contained in the sub-section.
 * @param g It is the resulting array that contains the value of the computed projection attenuation for each pixel.
 * @param isFirst Tells if `g` array is uninitialized or it is not, this function initializes it if necessary.
 * @return __global__ Indicates that this is a CUDA kernel function, so it is executed on the device (GPU) and not the host (CPU).
 */
__global__ void computeProjections(const unsigned short slice, const unsigned short nTheta, const unsigned nSidePixels, const double *const f, double *const g, const char isFirst) {
    __shared__ double l_gMins[BLKDIM];
    __shared__ double l_gMaxs[BLKDIM];
    const unsigned l_index = threadIdx.x + threadIdx.y * blockDim.x;
    const unsigned r = threadIdx.y + blockIdx.y * blockDim.y;
    const unsigned c = threadIdx.x + blockIdx.x * blockDim.x;
    l_gMins[l_index] = INFINITY;
    l_gMaxs[l_index] = -INFINITY;

    if (r < nSidePixels && c < nSidePixels) {
        double a[3][2];

        if (isFirst) {
            for (unsigned short positionIndex = 0; positionIndex < nTheta; positionIndex++) {
                g[positionIndex * nSidePixels * nSidePixels + r * nSidePixels + c] = 0.0;
            }
        }
        // Iterates over each source
        for (unsigned short positionIndex = 0; positionIndex < nTheta; positionIndex++) {
            const Point source = getSource(d_gl_sinTable, d_gl_cosTable, positionIndex);

            // Computes the attenuation over a single pixel of the detector
            const Point pixel = getPixel(d_gl_sinTable, d_gl_cosTable, r, c, positionIndex);

            // Computes Min-Max parametric values
            double aMin, aMax;
            double sidesPlanes[2];
            char isParallel = -1;
            getSidesXPlanes(sidesPlanes);
            if (!getIntersection(source.x, pixel.x, sidesPlanes, 2, &a[X][0])) {
                isParallel = X;
            }
            getSidesYPlanes(sidesPlanes, slice);
            if (!getIntersection(source.y, pixel.y, sidesPlanes, 2, &a[Y][0])) {
                isParallel = Y;
            }
            getSidesZPlanes(sidesPlanes);
            if (!getIntersection(source.z, pixel.z, sidesPlanes, 2, &a[Z][0])) {
                isParallel = Z;
            }

            aMin = getAMin(a, isParallel);
            aMax = getAMax(a, isParallel);

            if (aMin < aMax) {
                // Computes Min-Max plane indexes
                Ranges indices[3];
                indices[X] = getRangeOfIndex(source.x, pixel.x, isParallel, aMin, aMax, X);
                indices[Y] = getRangeOfIndex(source.y, pixel.y, isParallel, aMin, aMax, Y);
                indices[Z] = getRangeOfIndex(source.z, pixel.z, isParallel, aMin, aMax, Z);

                // Computes lengths of the arrays containing parametric value of the intersection with each set of parallel planes
                const unsigned short lenX = max(0, indices[X].maxIndx - indices[X].minIndx);
                const unsigned short lenY = max(0, indices[Y].maxIndx - indices[Y].minIndx);
                const unsigned short lenZ = max(0, indices[Z].maxIndx - indices[Z].minIndx);

                // Computes ray-planes intersection Nx + Ny + Nz
                double aX[MAX_PLANES];
                double aY[MAX_PLANES];
                double aZ[MAX_PLANES];
                getAllIntersections(source.x, pixel.x, indices[X], aX, X);
                getAllIntersections(source.y, pixel.y, indices[Y], aY, Y);
                getAllIntersections(source.z, pixel.z, indices[Z], aZ, Z);

                // Computes segments Nx + Ny + Nz
                double aMerged[MAX_PLANES_x3];
                const unsigned short lenA = merge3(aX, aY, aZ, lenX, lenY, lenZ, aMerged);

                // Associates each segment to the respective voxel Nx + Ny + Nz
                const unsigned pixelIndex = positionIndex * nSidePixels * nSidePixels + r * nSidePixels + c;
                g[pixelIndex] += computeAbsorption(slice, source, pixel, aMerged, lenA, f);
                l_gMins[l_index] = fmin(l_gMins[l_index], g[pixelIndex]);
                l_gMaxs[l_index] = fmax(l_gMaxs[l_index], g[pixelIndex]);
            }
        }
    }

    unsigned b_size = blockDim.x / 2;
    __syncthreads();
    while (b_size > 0) {
        if (l_index < b_size) {
            if (l_gMins[l_index] > l_gMins[l_index + b_size]) {
                l_gMins[l_index] = l_gMins[l_index + b_size];
            }
            if (l_gMaxs[l_index] < l_gMaxs[l_index + b_size]) {
                l_gMaxs[l_index] = l_gMaxs[l_index + b_size];
            }
        }
        b_size = b_size / 2;
        __syncthreads();
    }

    if (l_index == 0) {
        atomicMinDouble(&d_gMin, l_gMins[l_index]);
        atomicMaxDouble(&d_gMax, l_gMaxs[l_index]);
    }
}

#ifdef DEBUG
static void printSizeMaxGB(const char *name, size_t size, const char* type="") {
    if (size > 1024) {
        double approximation = size / 1024.0;
        if (approximation > 1024.0) {
            approximation = approximation / 1024.0;
            if (approximation > 1024.0) {
                approximation = approximation / 1024.0;
                printf("%s %s = %.3lf GB\n", type, name, approximation);
            } else {
                printf("%s %s = %.2lf MB\n", type, name, approximation);
            }
        } else {
            printf("%s %s = %.1lf KB\n", type, name, approximation);
        }
    } else {
        printf("%s %s = %lu B\n", type, name, size);
    }
}
#endif

/**
 * @brief Releases allocated resources of the CUDA environment.
 */
void termEnvironment(void) {
    cudaSafeCall(hipFree(d_f));
    cudaSafeCall(hipFree(d_g));
    free(gl_sinTable);
    free(gl_cosTable);
}

/**
 * @brief Collets GPU computed data on the CPU and releases allocated resources of the CUDA environment.
 *
 * @param g It is the resulting array that contains the value of the computed projection attenuation for each pixel.
 * @param sizeG It is the size of the output array that contains the value of the computed projection attenuation for each pixel.
 * @param gMin It is the minimum attenuation computed.
 * @param gMax It is the maximum attenuation computed.
 */
void termEnvironment(double *g, size_t sizeG, double *gMin, double *gMax) {
    cudaSafeCall(hipMemcpy(g, d_g, sizeG, hipMemcpyDeviceToHost));
    cudaSafeCall(hipMemcpyFromSymbol(gMin, HIP_SYMBOL(d_gMin), sizeof(d_gMin)));
    cudaSafeCall(hipMemcpyFromSymbol(gMax, HIP_SYMBOL(d_gMax), sizeof(d_gMax)));
    termEnvironment();
}

/**
 * @brief Allocates resources in the CUDA environment.
 *
 * @param sizeF It is the size of the input array that contains the coefficients of attenuation of the voxels contained in the
 * sub-section.
 * @param sizeG It is the size of the output array that contains the value of the computed projection attenuation for each pixel.
 * @param nTheta It is the number of angular positions.
 * @param nSidePixels It is the number of pixels per size of the detector.
 * @param gMin It is the minimum attenuation computed.
 * @param gMax It is the maximum attenuation computed.
 */
void initEnvironment(size_t *sizeF, size_t sizeG, const unsigned short nTheta, const unsigned nSidePixels, double *gMin, double *gMax) {
#ifdef DEBUG
    printf("CONFIG (threads and blocks):\n");
    const unsigned short tmp = (nSidePixels + BLKDIM_STEP - 1) / BLKDIM_STEP;
    printf("%s = %dx%d\n", "2D grid", tmp, tmp);
    printf("%s = %dx%d\n\n", "2D block", BLKDIM_STEP, BLKDIM_STEP);
    printf("%s = %d\n", "N blocks", tmp * tmp);
    printf("%s = %d\n\n", "N threads", BLKDIM_STEP * BLKDIM_STEP);
#endif
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_pixelDim), &gl_pixelDim, sizeof(d_pixelDim)));
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_angularTrajectory), &gl_angularTrajectory, sizeof(d_angularTrajectory)));
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_positionsAngularDistance), &gl_positionsAngularDistance, sizeof(d_positionsAngularDistance)));
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_objectSideLength), &gl_objectSideLength, sizeof(d_objectSideLength)));
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_detectorSideLength), &gl_detectorSideLength, sizeof(d_detectorSideLength)));
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_distanceObjectDetector), &gl_distanceObjectDetector, sizeof(d_distanceObjectDetector)));
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_distanceObjectSource), &gl_distanceObjectSource, sizeof(d_distanceObjectSource)));
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_voxelXDim), &gl_voxelXDim, sizeof(d_voxelXDim)));
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_voxelYDim), &gl_voxelYDim, sizeof(d_voxelYDim)));
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_voxelZDim), &gl_voxelZDim, sizeof(d_voxelZDim)));
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_nVoxel), gl_nVoxel, sizeof(d_nVoxel)));
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_nPlanes), gl_nPlanes, sizeof(d_nPlanes)));

    gl_sinTable = (double *) malloc(sizeof(double) * nTheta);
    gl_cosTable = (double *) malloc(sizeof(double) * nTheta);
    initTables(gl_sinTable, gl_cosTable, nTheta);
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_gl_sinTable), gl_sinTable, sizeof(d_gl_sinTable)));
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_gl_cosTable), gl_cosTable, sizeof(d_gl_cosTable)));

    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_gMin), gMin, sizeof(d_gMin)));
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_gMax), gMax, sizeof(d_gMax)));

    cudaSafeCall(hipMalloc((void **) &d_g, sizeG));

    if (!yVoxels) {
        size_t freeMem, totalMem;
        hipMemGetInfo(&freeMem, &totalMem);
        // At least 1 GB of estimated free global memory are necessary for the kernel execution in a 8 GB RAM GPU
        freeMem -= (totalMem * 2 / 8);
        unsigned voxelsY = gl_nVoxel[Y];
        size_t size = sizeof(double) * gl_nVoxel[X] * voxelsY * gl_nVoxel[Z];
        while (size > freeMem) {
            // 5 / 8 is around 5 GB if considering an 8 GB input size
            voxelsY = voxelsY * 5 / 8;
            if (voxelsY <= 0) {
                fprintf(stderr, "The voxels Y size is too small respect to the other sizes:\n"
                                "- N voxels X: %u.\n"
                                "- N voxels Y: %u.\n"
                                "- N voxels Z: %u.\n"
                                "Total size reduced to the minimum possible is %lu Bytes!\n"
                                "This is too much for this GPU with %lu Bytes of usable global memory (of %lu Bytes total)!\n",
                                gl_nVoxel[X], gl_nVoxel[Y], gl_nVoxel[Z], size, freeMem, totalMem);
                termEnvironment();
                exit(EXIT_FAILURE);
            }
            size = sizeof(double) * gl_nVoxel[X] * voxelsY * gl_nVoxel[Z];
        }
        yVoxels = voxelsY;
        *sizeF = size;
    } else {
        *sizeF = sizeof(double) * gl_nVoxel[X] * yVoxels * gl_nVoxel[Z];
    }

    cudaSafeCall(hipMalloc((void **) &d_f, *sizeF));
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_yVoxels), &yVoxels, sizeof(d_yVoxels)));
#ifdef DEBUG
    printSizeMaxGB("f", *sizeF, "GLOBAL");
    printSizeMaxGB("g", sizeG, "GLOBAL");
    printf("\n");
#endif
}

/**
 * @brief Computes the projection of a sub-section of the object into the detector for each source position.
 *
 * @param slice It is a number that indicates the first voxel in the y axis from which the projection is being computed.
 * @param f It is an array that contains the coefficients of attenuation of the voxels contained in the sub-section.
 * @param sizeF It is the size of the `f`.
 * @param nTheta It is the number of angular positions.
 * @param nSidePixels It is the number of pixels per size of the detector.
 * @param isFirst Tells if `g` array is uninitialized or it is not, this function tells to initialize it if necessary.
 */
void getProjections(const unsigned short slice, double *f, const size_t sizeF, const unsigned short nTheta, const unsigned nSidePixels, const char isFirst)
{
#ifdef PRINT
    printf("%.4lf> Copying f...\n", hpc_gettime());
#endif
    cudaSafeCall(hipMemcpy(d_f, f, sizeF, hipMemcpyHostToDevice));
    static dim3 block(BLKDIM_STEP, BLKDIM_STEP);
    static dim3 grid((nSidePixels + BLKDIM_STEP - 1) / BLKDIM_STEP, (nSidePixels + BLKDIM_STEP - 1) / BLKDIM_STEP);
#ifdef PRINT
    printf("%.4lf> Executing kernel...\n", hpc_gettime());
#endif
    computeProjections<<<grid, block>>>(slice, nTheta, nSidePixels, d_f, d_g, isFirst);
    cudaCheckError();
}

/**
 * @brief Reads the environment values used to compute the voxel grid from the specified binary file.
 *
 * @param filePointer It is the file pointer to read the values from.
 * @return EXIT_FAILURE in case of reading failure, EXIT_SUCCESS otherwise.
 */
int readSetUP(FILE *const filePointer)
{
    unsigned short buffer0[12];
    if (!fread(buffer0, sizeof(unsigned short), sizeof(buffer0) / sizeof(unsigned short), filePointer)) {
        return EXIT_FAILURE;
    }
    unsigned char i = 0;
    gl_pixelDim = buffer0[i++];
    gl_angularTrajectory = buffer0[i++];
    gl_positionsAngularDistance = buffer0[i++];
    gl_voxelXDim = buffer0[i++];
    gl_voxelYDim = buffer0[i++];
    gl_voxelZDim = buffer0[i++];
    gl_nVoxel[X] = buffer0[i++];
    gl_nVoxel[Y] = buffer0[i++];
    gl_nVoxel[Z] = buffer0[i++];
    gl_nPlanes[X] = buffer0[i++];
    gl_nPlanes[Y] = buffer0[i++];
    gl_nPlanes[Z] = buffer0[i];

    unsigned buffer1[4];
    if (!fread(buffer1, sizeof(unsigned), sizeof(buffer1) / sizeof(unsigned), filePointer)) {
        return EXIT_FAILURE;
    }
    i = 0;
    gl_objectSideLength = buffer1[i++];
    gl_detectorSideLength = buffer1[i++];
    gl_distanceObjectDetector = buffer1[i++];
    gl_distanceObjectSource = buffer1[i];

#ifdef PRINT_VARIABLES
    printf("Variables READ:\n");
    printf("- unsigned short:\n");
    printf("    gl_pixelDim = %hu\n", gl_pixelDim);
    printf("    gl_angularTrajectory = %hu\n", gl_angularTrajectory);
    printf("    gl_positionsAngularDistance = %hu\n", gl_positionsAngularDistance);
    printf("    gl_voxelXDim = %hu\n", gl_voxelXDim);
    printf("    gl_voxelYDim = %hu\n", gl_voxelYDim);
    printf("    gl_voxelZDim = %hu\n", gl_voxelZDim);
    printf("    gl_nVoxel[X] = %hu\n", gl_nVoxel[X]);
    printf("    gl_nVoxel[Y] = %hu\n", gl_nVoxel[Y]);
    printf("    gl_nVoxel[Z] = %hu\n", gl_nVoxel[Z]);
    printf("    gl_nPlanes[X] = %hu\n", gl_nPlanes[X]);
    printf("    gl_nPlanes[Y] = %hu\n", gl_nPlanes[Y]);
    printf("    gl_nPlanes[Z] = %hu\n", gl_nPlanes[Z]);
    printf("- unsigned:\n");
    printf("    gl_objectSideLength = %u\n", gl_objectSideLength);
    printf("    gl_detectorSideLength = %u\n", gl_detectorSideLength);
    printf("    gl_distanceObjectDetector = %u\n", gl_distanceObjectDetector);
    printf("    gl_distanceObjectSource = %u\n", gl_distanceObjectSource);
#endif

    return EXIT_SUCCESS;
}

int main(int argc, char *argv[])
{
    if (argc < 2 || argc > 4) {
        fprintf(stderr, "Usage: %s INPUT [OUTPUT] [Y_MAX_VOXELS]\n"
                        "- INPUT: The first parameter is the name of the input file.\n"
                        "- [OUTPUT]: The second parameter is the name of a .pgm file to store the output at.\n"
                        "- [Y_MAX_VOXELS]: The third parameter is the maximum number of voxels considered in the Y axis for each iteration.\n",
                        argv[0]);
        return EXIT_FAILURE;
    }
    const char *const inputFileName = argv[1];
    const char *outputFileName = NULL;
    if (argc > 2) {
        outputFileName = argv[2];
    }

    FILE *const inputFilePointer = fopen(inputFileName, "rb");
    if (!inputFilePointer) {
        fprintf(stderr, "Unable to open file '%s'!\n", inputFileName);
        return EXIT_FAILURE;
    }

    if (readSetUP(inputFilePointer) == EXIT_FAILURE) {
        fprintf(stderr, "Unable to read from file '%s'!\n", inputFileName);
        return EXIT_FAILURE;
    }
    if (gl_nPlanes[X] > MAX_PLANES || gl_nPlanes[Y] > MAX_PLANES || gl_nPlanes[Z] > MAX_PLANES) {
        fprintf(stderr, "The maximum number of planes per axis is %u planes!\n", MAX_PLANES);
        return EXIT_FAILURE;
    }
    if (argc > 3) {
        const int yMaxVoxels = atoi(argv[3]);
        yVoxels = min(gl_nVoxel[Y], max(yVoxels, 1));
    }
    // Number of angular positions
    double partialTime = hpc_gettime();
    const unsigned short nTheta = gl_angularTrajectory / gl_positionsAngularDistance + 1;
    if (nTheta > MAX_TABLES_SIZE) {
        fprintf(stderr, "Number of positions required %u is too large, max %lu!\n", nTheta, MAX_TABLES_SIZE);
        exit(EXIT_FAILURE);
    }
    const unsigned nSidePixels = gl_detectorSideLength / gl_pixelDim;
    // Size of the array containing the computed attenuation detected in each pixel of the detector
    const size_t sizeG = sizeof(double) * nSidePixels * nSidePixels * nTheta;
    // Minimum and maximum attenuation computed
    double gMinValue = INFINITY, gMaxValue = -INFINITY;
    size_t sizeF;
    initEnvironment(&sizeF, sizeG, nTheta, nSidePixels, &gMinValue, &gMaxValue);
    // Array containing the coefficients of each voxel
    double *const f = (double *) malloc(sizeF);
    double totalTime = hpc_gettime() - partialTime;

    // Iterates over object subsections
    for (unsigned short slice = 0; slice < gl_nVoxel[Y]; slice += yVoxels) {
        unsigned short nOfSlices;

        if (gl_nVoxel[Y] - slice < yVoxels) {
            nOfSlices = gl_nVoxel[Y] - slice;
        } else {
            nOfSlices = yVoxels;
        }

        // Read voxels coefficients
#ifdef PRINT
        printf("%.4lf> Reading f...\n", hpc_gettime());
#endif
        if (!fread(f, sizeof(double), (size_t) gl_nVoxel[X] * nOfSlices * gl_nVoxel[Z], inputFilePointer)) {
            fprintf(stderr, "Unable to read from file '%s'!\n", inputFileName);
            free(f);
            termEnvironment();
            return EXIT_FAILURE;
        }

#ifdef PRINT
        static unsigned short it = 0;
        printf("%.4lf> IT %u of size %hu\n", hpc_gettime(), ++it, nOfSlices);
#endif
        // Computes subsection projection
        partialTime = hpc_gettime();
        getProjections(slice, f, sizeF, nTheta, nSidePixels, !slice);
        totalTime += hpc_gettime() - partialTime;
    }
    fclose(inputFilePointer);
    partialTime = hpc_gettime();
    free(f);
    // Array containing the computed attenuation detected in each pixel of the detector
    double *const g = (double *) malloc(sizeG);
    termEnvironment(g, sizeG, &gMinValue, &gMaxValue);
    totalTime += hpc_gettime() - partialTime;
    printf("Execution time (s) %.2f\n", totalTime);

    if (outputFileName != NULL) {
        FILE *const outputFilePointer = fopen(outputFileName, "w");
        if (!outputFileName) {
            fprintf(stderr, "Unable to open file '%s'!\n", outputFileName);
            free(g);
            return EXIT_FAILURE;
        }
        // Iterates over each attenuation value computed, prints a value between [0-255]
        fprintf(outputFilePointer, "P2\n%d %d\n255", nSidePixels, nSidePixels * nTheta);
        for (unsigned short positionIndex = 0; positionIndex < nTheta; positionIndex++) {
            const double angle = -(double) gl_angularTrajectory / 2 + (double) positionIndex * gl_positionsAngularDistance;
            fprintf(outputFilePointer, "\n#%lf", angle);
            for (unsigned i = 0; i < nSidePixels; i++) {
                fprintf(outputFilePointer, "\n");
                for (unsigned j = 0; j < nSidePixels; j++) {
                    const unsigned pixelIndex = positionIndex * nSidePixels * nSidePixels + i * nSidePixels + j;
                    const int color = (g[pixelIndex] - gMinValue) * 255 / (gMaxValue - gMinValue);
                    fprintf(outputFilePointer, "%d ", color);
                }
            }
        }
        fclose(outputFilePointer);
    }
    free(g);

    return EXIT_SUCCESS;
}
